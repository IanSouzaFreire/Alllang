// Cuda-C++

#include <hip/hip_runtime.h>

struct Vec2 {
  double x, y;

  __host__ __device__ Vec2(double X, double Y) noexcept
    : x(X), y(Y) {}

  __host__ __device__ Vec2& add(const Vec2& other) noexcept {
    x += other.x;
    y += other.y;
    return *this;
  }

  __host__ __device__ Vec2& scale(double scalar) noexcept {
    x *= scalar;
    y *= scalar;
    return *this;
  }
};

struct Size {
  double width, height;

  __host__ __device__ Size(double w, double h) noexcept
    : width(w), height(h) {}
};

struct GameObject {
  Vec2 position;
  Size size;

  __host__ __device__ GameObject(const Vec2& pos, const Size& sz) noexcept
    : position(pos), size(sz) {}

  __host__ __device__ bool horizontalCollision(const GameObject& other) const noexcept {
    return (other.position.x < position.x + size.width &&
            other.position.x + other.size.width > position.x);
  }

  __host__ __device__ bool verticalCollision(const GameObject& other) const noexcept {
    return (other.position.y < position.y + size.height &&
            other.position.y + other.size.height > position.y);
  }

  __host__ __device__ bool collide(const GameObject& other) const noexcept {
    return horizontalCollision(other) && verticalCollision(other);
  }
};

struct MovableObject : public GameObject {
  Vec2 velocity;
  Vec2 acceleration;

  __host__ __device__ MovableObject(const Vec2& pos, const Size& sz, const Vec2& vel, const Vec2& acc) noexcept
    : GameObject(pos, sz), velocity(vel), acceleration(acc) {}

  __host__ __device__ void update(double deltaTime) noexcept {
    velocity.add(acceleration.scale(deltaTime));
    position.add(velocity.scale(deltaTime));
  }
};

struct Car : public MovableObject {
  double turnAngle;

  __host__ __device__ Car(const Vec2& pos, const Size& sz, const Vec2& vel, const Vec2& acc, double angle) noexcept
    : MovableObject(pos, sz, vel, acc), turnAngle(angle) {}

  __host__ __device__ void turn(double angle) noexcept {
    turnAngle += angle;
  }

  __host__ __device__ void update(double deltaTime) noexcept {
    MovableObject::update(deltaTime);
  }
};
